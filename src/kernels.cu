#include "hip/hip_runtime.h"

#include <stdio.h>

__global__
void hello(float *v1, float *v2, float *vout, int len) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < len) {
    vout[i] = v1[i] + v2[i];
  }
}